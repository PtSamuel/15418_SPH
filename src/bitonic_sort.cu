#include "hip/hip_runtime.h"
#include "Particle.h"
#include "bitonic_sort.h"

#include <stdio.h>

#define BLOCK_DIM 16
// DO NOT DO THIS!!!
// #define THREADS_PER_BLOCK BLOCK_DIM * BLOCK_DIM
#define THREADS_PER_BLOCK (BLOCK_DIM * BLOCK_DIM)

__device__ void cas(Particle *p1, Particle *p2, int polarity) {
    bool misordered = p1->block > p2->block || (p1->block == p2->block && p1->id > p2->id);
    if(!polarity) {
        if(misordered) {
            Particle temp = *p1;
            *p1 = *p2;
            *p2 = temp;
        }
    } else {
        if(!misordered) {
            Particle temp = *p1;
            *p1 = *p2;
            *p2 = temp;
        }
    }
}

__global__ void compare_and_swap(Particle *particles, int n, int stride, int groupsize) {
    int index = blockIdx.x * THREADS_PER_BLOCK + threadIdx.y * BLOCK_DIM + threadIdx.x;

    if(index >= n / 2) return;

    int groupid = index / groupsize;
    int groupstart = groupsize * 2 * groupid;
    int groupsubid = index - groupid * groupsize;
    int groupmatesubid = groupsubid + groupsize;

    int sort_order = (groupstart / stride) % 2;

    printf("thread index: %d, groupstart: %d, subid: %d, matesubid: %d, order: %d\n", index, groupstart, groupsubid, groupmatesubid, sort_order);

    // particles[groupstart + groupsubid].block = 0;
    // particles[groupstart + groupmatesubid].block = 0;

    cas(&particles[groupstart + groupsubid], &particles[groupstart + groupmatesubid], sort_order);
}

void bitonic_sort(Particle *p, int n) {
    Particle *particles;
    hipMalloc(&particles, sizeof(Particle) * n);
    hipMemcpy(particles, p, sizeof(Particle) * n, hipMemcpyHostToDevice);

    int num_tasks = n / 2;

    int num_blocks = (num_tasks + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    dim3 grid_dim(num_blocks, 1);
    dim3 block_dim(BLOCK_DIM, BLOCK_DIM);
    
    int stage = 1;
    for(int stride = 2; stride <= n; stride *= 2) {
        for(int groupsize = stage; groupsize >= 1; groupsize /= 2) {
            printf("stride: %d, groupsize: %d\n", stride, groupsize);
            compare_and_swap<<<grid_dim, block_dim>>>(particles, n, stride, groupsize);
            hipDeviceSynchronize();
        }
        stage *= 2;
    }

    hipMemcpy(p, particles, sizeof(Particle) * n, hipMemcpyDeviceToHost);

    for(int i = 0; i < n; i++) {
        printf("%d: %d\n", p[i].id, p[i].block);
    }
}